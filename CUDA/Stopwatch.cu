#include "Stopwatch.cuh"

#include <iostream>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

Stopwatch::Stopwatch()
{
    HANDLE_ERROR(hipEventCreate(&startTime));
    HANDLE_ERROR(hipEventCreate(&stopTime));
}

void Stopwatch::start()
{
    HANDLE_ERROR(hipEventRecord(startTime, 0));
}

void Stopwatch::stop()
{
    HANDLE_ERROR(hipEventRecord(stopTime, 0));
    HANDLE_ERROR(hipEventSynchronize(stopTime));
}

void Stopwatch::displayTime()
{
    HANDLE_ERROR(hipEventElapsedTime(&time, startTime, stopTime));
    std::cout << "Elapsed time: " << time << " ms" << std::endl;
}
